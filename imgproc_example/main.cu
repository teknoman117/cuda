#include "hip/hip_runtime.h"
//
//  imgproc_main.cpp
//  
//
//  Created by Nathaniel Lewis on 3/8/12.
//  Copyright (c) 2012 E1FTW Games. All rights reserved.
//

#include <iostream>
#include <string>

#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>

// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
 * Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
 * of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
 *
 * @param source      Source image host pinned memory pointer
 * @param width       Source image width
 * @param height      Source image height
 * @param paddingX    source image padding along x 
 * @param paddingY    source image padding along y
 * @param kOffset     offset into kernel store constant memory 
 * @param kWidth      kernel width
 * @param kHeight     kernel height
 * @param destination Destination image host pinned memory pointer
 */
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, ssize_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    float sum = 0.0;
    int   pWidth = kWidth/2;
    int   pHeight = kHeight/2;

    // Only execute for valid pixels
    if(x >= pWidth+paddingX &&
       y >= pHeight+paddingY &&
       x < (blockDim.x * gridDim.x)-pWidth-paddingX &&
       y < (blockDim.y * gridDim.y)-pHeight-paddingY)
    {
        for(int j = -pHeight; j <= pHeight; j++)
        {
            for(int i = -pWidth; i <= pWidth; i++)
            {
                // Sample the weight for this location
                int ki = (i+pWidth);
                int kj = (j+pHeight);
                float w  = convolutionKernelStore[(kj * kWidth) + ki + kOffset];

        
                sum += w * float(source[((y+j) * width) + (x+i)]);
            }
        }
    }
    
    // Average the sum
    destination[(y * width) + x] = (unsigned char) sum;
}

// converts the pythagoran theorem along a vector on the GPU
__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    float af = float(a[idx]);
    float bf = float(b[idx]);

    c[idx] = (unsigned char) sqrtf(af*af + bf*bf);
}

// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
unsigned char* createImageBuffer(unsigned int bytes, unsigned char **devicePtr)
{
    unsigned char *ptr = NULL;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
    hipHostGetDevicePointer(devicePtr, ptr, 0);
    return ptr;
}



int main (int argc, char** argv)
{
    // Open a webcamera
    cv::VideoCapture camera(0);
    cv::Mat          frame;
    if(!camera.isOpened()) 
        return -1;
    
    // Create the capture windows
    cv::namedWindow("Source");
    cv::namedWindow("Greyscale");
    cv::namedWindow("Blurred");
    cv::namedWindow("Sobel");

    // Create the cuda event timers 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Create the gaussian kernel (sum = 159)
    const float gaussianKernel5x5[25] = 
    {
        2.f/159.f,  4.f/159.f,  5.f/159.f,  4.f/159.f, 2.f/159.f,   
        4.f/159.f,  9.f/159.f, 12.f/159.f,  9.f/159.f, 4.f/159.f,   
        5.f/159.f, 12.f/159.f, 15.f/159.f, 12.f/159.f, 5.f/159.f,   
        4.f/159.f,  9.f/159.f, 12.f/159.f,  9.f/159.f, 4.f/159.f,   
        2.f/159.f,  4.f/159.f,  5.f/159.f,  4.f/159.f, 2.f/159.f,   
    };
    hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), gaussianKernel5x5, sizeof(gaussianKernel5x5), 0);
    const ssize_t gaussianKernel5x5Offset = 0;

    // Sobel gradient kernels
    const float sobelGradientX[9] =
    {
        -1.f, 0.f, 1.f,
        -2.f, 0.f, 2.f,
        -1.f, 0.f, 1.f,
    };
    const float sobelGradientY[9] =
    {
        1.f, 2.f, 1.f,
        0.f, 0.f, 0.f,
        -1.f, -2.f, -1.f,
    };
    hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), sobelGradientX, sizeof(sobelGradientX), sizeof(gaussianKernel5x5));
    hipMemcpyToSymbol(HIP_SYMBOL(convolutionKernelStore), sobelGradientY, sizeof(sobelGradientY), sizeof(gaussianKernel5x5) + sizeof(sobelGradientX));
    const ssize_t sobelGradientXOffset = sizeof(gaussianKernel5x5)/sizeof(float);
    const ssize_t sobelGradientYOffset = sizeof(sobelGradientX)/sizeof(float) + sobelGradientXOffset;
 
    // Create CPU/GPU shared images - one for the initial and one for the result
    camera >> frame;
    unsigned char *sourceDataDevice, *blurredDataDevice, *edgesDataDevice;
    cv::Mat source  (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &sourceDataDevice));
    cv::Mat blurred (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &blurredDataDevice));
    cv::Mat edges   (frame.size(), CV_8U, createImageBuffer(frame.size().width * frame.size().height, &edgesDataDevice));

    // Create two temporary images (for holding sobel gradients)
    unsigned char *deviceGradientX, *deviceGradientY;
    hipMalloc(&deviceGradientX, frame.size().width * frame.size().height);
    hipMalloc(&deviceGradientY, frame.size().width * frame.size().height);
    
    // Loop while capturing images
    while(1)
    {
        // Capture the image and store a gray conversion to the gpu
        camera >> frame;
        cv::cvtColor(frame, source, cv::COLOR_BGR2GRAY);
        
        // Record the time it takes to process
        hipEventRecord(start);
        {
            // convolution kernel launch parameters
            dim3 cblocks (frame.size().width / 16, frame.size().height / 16);
            dim3 cthreads(16, 16);

            // pythagoran kernel launch paramters
            dim3 pblocks (frame.size().width * frame.size().height / 256);
            dim3 pthreads(256, 1);

            // Perform the gaussian blur (first kernel in store @ 0)
            convolve<<<cblocks,cthreads>>>(sourceDataDevice, frame.size().width, frame.size().height, 0, 0, gaussianKernel5x5Offset, 5, 5, blurredDataDevice);

            // Perform the sobel gradient convolutions (x&y padding is now 2 because there is a border of 2 around a 5x5 gaussian filtered image)
            convolve<<<cblocks,cthreads>>>(blurredDataDevice, frame.size().width, frame.size().height, 2, 2, sobelGradientXOffset, 3, 3, deviceGradientX);
            convolve<<<cblocks,cthreads>>>(blurredDataDevice, frame.size().width, frame.size().height, 2, 2, sobelGradientYOffset, 3, 3, deviceGradientY);
            pythagoras<<<pblocks,pthreads>>>(deviceGradientX, deviceGradientY, edgesDataDevice);

            hipDeviceSynchronize();
        }
        hipEventRecord(stop);
        
        // Display the elapsed time
        float ms = 0.0f;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        std::cout << "Elapsed GPU time: " << ms << " milliseconds" << std::endl;

        // Show the results
        cv::imshow("Source", frame);
        cv::imshow("Greyscale", source);
        cv::imshow("Blurred", blurred);
        cv::imshow("Sobel", edges);

        // Spin
        if(cv::waitKey(1) == 27) break;
    }
    
    // Exit
    hipHostFree(source.data);
    hipHostFree(blurred.data);
    hipHostFree(edges.data);
    hipFree(deviceGradientX);
    hipFree(deviceGradientY);

    return 0;
}
